#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <string>
#include <iostream>
#include <chrono>
#include <thread>
#include <fstream>

using namespace std;

constexpr uint32_t d0[256] = {
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x000000f8, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x000000fc,
0x000000d0, 0x000000d4, 0x000000d8, 0x000000dc, 0x000000e0, 0x000000e4,
0x000000e8, 0x000000ec, 0x000000f0, 0x000000f4, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x00000000,
0x00000004, 0x00000008, 0x0000000c, 0x00000010, 0x00000014, 0x00000018,
0x0000001c, 0x00000020, 0x00000024, 0x00000028, 0x0000002c, 0x00000030,
0x00000034, 0x00000038, 0x0000003c, 0x00000040, 0x00000044, 0x00000048,
0x0000004c, 0x00000050, 0x00000054, 0x00000058, 0x0000005c, 0x00000060,
0x00000064, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x00000068, 0x0000006c, 0x00000070, 0x00000074, 0x00000078,
0x0000007c, 0x00000080, 0x00000084, 0x00000088, 0x0000008c, 0x00000090,
0x00000094, 0x00000098, 0x0000009c, 0x000000a0, 0x000000a4, 0x000000a8,
0x000000ac, 0x000000b0, 0x000000b4, 0x000000b8, 0x000000bc, 0x000000c0,
0x000000c4, 0x000000c8, 0x000000cc, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff
};
constexpr uint32_t d1[256] = {
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x0000e003, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x0000f003,
0x00004003, 0x00005003, 0x00006003, 0x00007003, 0x00008003, 0x00009003,
0x0000a003, 0x0000b003, 0x0000c003, 0x0000d003, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x00000000,
0x00001000, 0x00002000, 0x00003000, 0x00004000, 0x00005000, 0x00006000,
0x00007000, 0x00008000, 0x00009000, 0x0000a000, 0x0000b000, 0x0000c000,
0x0000d000, 0x0000e000, 0x0000f000, 0x00000001, 0x00001001, 0x00002001,
0x00003001, 0x00004001, 0x00005001, 0x00006001, 0x00007001, 0x00008001,
0x00009001, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x0000a001, 0x0000b001, 0x0000c001, 0x0000d001, 0x0000e001,
0x0000f001, 0x00000002, 0x00001002, 0x00002002, 0x00003002, 0x00004002,
0x00005002, 0x00006002, 0x00007002, 0x00008002, 0x00009002, 0x0000a002,
0x0000b002, 0x0000c002, 0x0000d002, 0x0000e002, 0x0000f002, 0x00000003,
0x00001003, 0x00002003, 0x00003003, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff
};
constexpr uint32_t d2[256] = {
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x00800f00, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x00c00f00,
0x00000d00, 0x00400d00, 0x00800d00, 0x00c00d00, 0x00000e00, 0x00400e00,
0x00800e00, 0x00c00e00, 0x00000f00, 0x00400f00, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x00000000,
0x00400000, 0x00800000, 0x00c00000, 0x00000100, 0x00400100, 0x00800100,
0x00c00100, 0x00000200, 0x00400200, 0x00800200, 0x00c00200, 0x00000300,
0x00400300, 0x00800300, 0x00c00300, 0x00000400, 0x00400400, 0x00800400,
0x00c00400, 0x00000500, 0x00400500, 0x00800500, 0x00c00500, 0x00000600,
0x00400600, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x00800600, 0x00c00600, 0x00000700, 0x00400700, 0x00800700,
0x00c00700, 0x00000800, 0x00400800, 0x00800800, 0x00c00800, 0x00000900,
0x00400900, 0x00800900, 0x00c00900, 0x00000a00, 0x00400a00, 0x00800a00,
0x00c00a00, 0x00000b00, 0x00400b00, 0x00800b00, 0x00c00b00, 0x00000c00,
0x00400c00, 0x00800c00, 0x00c00c00, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff
};
constexpr uint32_t d3[256] = {
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x003e0000, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x003f0000,
0x00340000, 0x00350000, 0x00360000, 0x00370000, 0x00380000, 0x00390000,
0x003a0000, 0x003b0000, 0x003c0000, 0x003d0000, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x00000000,
0x00010000, 0x00020000, 0x00030000, 0x00040000, 0x00050000, 0x00060000,
0x00070000, 0x00080000, 0x00090000, 0x000a0000, 0x000b0000, 0x000c0000,
0x000d0000, 0x000e0000, 0x000f0000, 0x00100000, 0x00110000, 0x00120000,
0x00130000, 0x00140000, 0x00150000, 0x00160000, 0x00170000, 0x00180000,
0x00190000, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x001a0000, 0x001b0000, 0x001c0000, 0x001d0000, 0x001e0000,
0x001f0000, 0x00200000, 0x00210000, 0x00220000, 0x00230000, 0x00240000,
0x00250000, 0x00260000, 0x00270000, 0x00280000, 0x00290000, 0x002a0000,
0x002b0000, 0x002c0000, 0x002d0000, 0x002e0000, 0x002f0000, 0x00300000,
0x00310000, 0x00320000, 0x00330000, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff,
0x01ffffff, 0x01ffffff, 0x01ffffff, 0x01ffffff
};

__device__ constexpr unsigned char unb64[256] = {
  0,   0,   0,   0,   0,   0,   0,   0,   0,   0, //10 
  0,   0,   0,   0,   0,   0,   0,   0,   0,   0, //20 
  0,   0,   0,   0,   0,   0,   0,   0,   0,   0, //30 
  0,   0,   0,   0,   0,   0,   0,   0,   0,   0, //40 
  0,   0,   0,  62,   0,   0,   0,  63,  52,  53, //50 
 54,  55,  56,  57,  58,  59,  60,  61,   0,   0, //60 
  0,   0,   0,   0,   0,   0,   1,   2,   3,   4, //70 
  5,   6,   7,   8,   9,  10,  11,  12,  13,  14, //80 
 15,  16,  17,  18,  19,  20,  21,  22,  23,  24, //90 
 25,   0,   0,   0,   0,   0,   0,  26,  27,  28, //100 
 29,  30,  31,  32,  33,  34,  35,  36,  37,  38, //110 
 39,  40,  41,  42,  43,  44,  45,  46,  47,  48, //120 
 49,  50,  51,   0,   0,   0,   0,   0,   0,   0, //130 
  0,   0,   0,   0,   0,   0,   0,   0,   0,   0, //140 
  0,   0,   0,   0,   0,   0,   0,   0,   0,   0, //150 
  0,   0,   0,   0,   0,   0,   0,   0,   0,   0, //160 
  0,   0,   0,   0,   0,   0,   0,   0,   0,   0, //170 
  0,   0,   0,   0,   0,   0,   0,   0,   0,   0, //180 
  0,   0,   0,   0,   0,   0,   0,   0,   0,   0, //190 
  0,   0,   0,   0,   0,   0,   0,   0,   0,   0, //200 
  0,   0,   0,   0,   0,   0,   0,   0,   0,   0, //210 
  0,   0,   0,   0,   0,   0,   0,   0,   0,   0, //220 
  0,   0,   0,   0,   0,   0,   0,   0,   0,   0, //230 
  0,   0,   0,   0,   0,   0,   0,   0,   0,   0, //240 
  0,   0,   0,   0,   0,   0,   0,   0,   0,   0, //250 
  0,   0,   0,   0,   0,   0,
};

#define CHARPAD 'A'
#define BADCHAR 0x01FFFFFF

int modp_b64_decode_h(char* dest, char* src, int len) {

	if (len < 4 || (len % 4 != 0)) return -1; /* error */
	/* there can be at most 2 pad chars at the end */
	if (src[len - 1] == CHARPAD) {
		len--;
		if (src[len - 1] == CHARPAD) {
			len--;
		}
	}

	int i;
	int leftover = len % 4;
	int chunks = (leftover == 0) ? len / 4 - 1 : len / 4;

	uint8_t* p = (uint8_t*)dest;
	uint32_t x = 0;
	uint32_t* destInt = (uint32_t*)p;
	uint32_t* srcInt = (uint32_t*)src;
	uint32_t y = *srcInt++;
	for (i = 0; i < chunks; ++i) {
		x = d0[y & 0xff] |
			d1[(y >> 8) & 0xff] |
			d2[(y >> 16) & 0xff] |
			d3[(y >> 24) & 0xff];

		if (x >= BADCHAR) return -1;
		*destInt = x;
		p += 3;
		destInt = (uint32_t*)p;
		y = *srcInt++;
	}


	switch (leftover) {
	case 0:
		x = d0[y & 0xff] |
			d1[(y >> 8) & 0xff] |
			d2[(y >> 16) & 0xff] |
			d3[(y >> 24) & 0xff];

		if (x >= BADCHAR) return -1;
		*p++ = ((uint8_t*)(&x))[0];
		*p++ = ((uint8_t*)(&x))[1];
		*p = ((uint8_t*)(&x))[2];
		return (chunks + 1) * 3;
		break;
	case 1:  /* with padding this is an impossible case */
		x = d0[y & 0xff];
		*p = *((uint8_t*)(&x)); // i.e. first char/byte in int
		break;
	case 2: // * case 2, 1  output byte */
		x = d0[y & 0xff] | d1[y >> 8 & 0xff];
		*p = *((uint8_t*)(&x)); // i.e. first char
		break;
	default: /* case 3, 2 output bytes */
		x = d0[y & 0xff] |
			d1[y >> 8 & 0xff] |
			d2[y >> 16 & 0xff];  /* 0x3c */
		*p++ = ((uint8_t*)(&x))[0];
		*p = ((uint8_t*)(&x))[1];
		break;
	}

	if (x >= BADCHAR) return -1;

	return 0;// 3 * chunks + (6 * leftover) / 8;
}

#define CIPHERTEXT_LEN 68
#define UTF8_LEN 50

//__device__ const char* CIPHERTEXT = "DePk6rqSKIcsDzx177WKCeD6uEYOo3iRkMszgy1sMJLD8rbSSP2J+FGF3L3yL8GmQQAA";
__device__ constexpr char CIPHERTEXTd[CIPHERTEXT_LEN] = {3,30,15,36,58,43,42,18,10,8,28,44,3,51,49,53,59,59,22,10,2,30,3,58,46,4,24,14,40,55,34,17,36,12,44,51,32,50,53,44,12,9,11,3,60,43,27,18,18,15,54,9,62,5,6,5,55,11,55,50,11,60,6,38,16,16,0,0};
__device__ constexpr char* ALPHABETd = "ABCDEFGHIJKLMNOPQRSTUVWXYZabcdefghijklmnopqrstuvwxyz0123456789+/";
constexpr char CIPHERTEXT[CIPHERTEXT_LEN] = {3,30,15,36,58,43,42,18,10,8,28,44,3,51,49,53,59,59,22,10,2,30,3,58,46,4,24,14,40,55,34,17,36,12,44,51,32,50,53,44,12,9,11,3,60,43,27,18,18,15,54,9,62,5,6,5,55,11,55,50,11,60,6,38,16,16,0,0};
constexpr char* ALPHABET = "ABCDEFGHIJKLMNOPQRSTUVWXYZabcdefghijklmnopqrstuvwxyz0123456789+/";
__device__ constexpr char* DECODE_TEST = "U0RG";

__global__ void compute(
	size_t key_length,
	size_t max_key_length,
	char* keys,
	char* decrypted_base64
) {
	size_t idx = threadIdx.x + blockDim.x * blockIdx.x;
	//printf("thread #%u is doing a thing\n", idx);

	char* base64_ptr = decrypted_base64 + idx * CIPHERTEXT_LEN;
	char* current = keys + idx * max_key_length;

	for (uint8_t i = 0; i < CIPHERTEXT_LEN; i++) {
		//*/
		char* out = base64_ptr + i;
		char new_ch = CIPHERTEXTd[i] - current[i % key_length];
		if (new_ch < 0) {
			*out = ALPHABETd[new_ch + 64];
		} else {
			*out = ALPHABETd[new_ch];
		}
		//*/
		
		//base64_ptr[i] = DECODE_TEST[i % 4];
	}
}

__global__ void decode_base64(char* decrypted_base64, char* decoded_utf8) {
	size_t idx = threadIdx.x + blockDim.x * blockIdx.x;

	char* base64_ptr = decrypted_base64 + idx * 4; //1 thread per 4 bytes
	char* utf8_ptr = decoded_utf8 + idx * 3;	   //1 thread per 3 bytes

	//int A = unb64[*base64_ptr];
	uint8_t B = unb64[base64_ptr[1]];
	uint8_t C = unb64[base64_ptr[2]];
	//int D = unb64[base64_ptr[3]];

	// Just unmap each sextet to THE NUMBER it represents.
	// You then have to pack it in bin,
	// we go in groups of 4 sextets, 
	// and pull out 3 octets per quad of sextets.
	//    bin[0]       bin[1]      bin[2]
	// +-----------+-----------+-----------+
	// | 0000 0011   0111 1011   1010 1101 |
	// +-AAAA AABB   BBBB CCCC   CCDD DDDD
	// or them
	*utf8_ptr = (/*A*/unb64[*base64_ptr] << 2) | (B >> 4); // OR in last 2 bits of B

	// The 2nd byte is the bottom 4 bits of B for the upper nibble,
	// and the top 4 bits of C for the lower nibble.
	utf8_ptr[1] = (B << 4) | (C >> 2);
	utf8_ptr[2] = (C << 6) | (unb64[base64_ptr[3]]);//(D); // shove C up to top 2 bits, or with D
}

__global__ void validate_utf8(size_t max_key_length, char* keys, char* decoded_utf8, char* decode_success) {
	size_t idx = threadIdx.x + blockDim.x * blockIdx.x;

	bool valid = true;
	char* utf8_ptr = decoded_utf8 + idx * UTF8_LEN;

	for (char* ptr = utf8_ptr + UTF8_LEN; ptr > utf8_ptr; --ptr) {
		uint8_t c = *ptr;
		if (c < 0x20 || c > 0x7E) {
			valid = false;
		}
	}

	if (valid) {
		char* key_ptr = keys + idx*max_key_length;
		char* success_ptr = decode_success + idx*max_key_length;
		for (uint8_t i = 0; i < max_key_length; ++i) {
			success_ptr[i] = ALPHABET[key_ptr[i]];
		}
	}
}

#define THREADS_PER_BLOCK 224

void add_one(char* ptr, size_t &len) {
	for (int i = 1;; ++i) {
		if (*ptr < 63) {(*ptr)++; return;}
		*ptr++ = 0;
		if (i == len) {len++; return;}
	}
}

void swap(void** ptr_a, void** ptr_b) {
	void* mid = *ptr_a;
	*ptr_a = *ptr_b;
	*ptr_b = mid;
}

int main(int argc, char *argv[]) {
	hipSetDeviceFlags(hipDeviceMapHost);
	size_t block_count = 89286;
	size_t max_key_size = 16;
	size_t current_length = 3;
	string current = string(max_key_size, 'A');
	char* current_ptr = &current[0];

	hipError_t cuda_err;
	int device_id = 0;

	if (argc > 1) {
		try {
			device_id = stoi(argv[1]);
		} catch (std::exception /*const &e*/) {
			//cout << " ERROR: invalid argument #2; number in range 0-" << UINT_MAX << " expected" << endl;
			//return 1;
		}

		try {
			block_count = stoi(argv[2]);
		} catch(std::exception /*const &e*/) {
			//cout << " ERROR: invalid argument #2; number in range 0-" << UINT_MAX << " expected" << endl;
			//return 1;
		}
		if (argc > 4 && argv[4] != NULL && argv[4][0] != '\0') {
			try {
				max_key_size = stoi(argv[3]);
				current = string('A', max_key_size);
			} catch(std::exception /*const &e*/) {
				//cout << " ERROR: invalid argument #4; number in range 0-" << UINT_MAX << " expected" << endl;
				//return 1;
			}
		}
		if (argc > 3 && argv[3] != NULL && argv[3][0] != '\0') {
			//cout << " ERROR: invalid argument #3; string expected" << endl;
			//return 1;
			current_length = 0;
			for (char* ch = argv[3];; ++ch) {
				if (*ch == '\0') {break;}
				current[current_length] = *ch;
				++current_length;
			}
		}
	}

	ofstream file;
	file.open("OUTPUT.txt");

	if (hipSetDevice(device_id) != hipSuccess) {
		cout << "invalid device ID, defaulting to 0";
		if (hipSetDevice(0) != hipSuccess) {
			cout << "hipSetDevice failed! Do you have a CUDA-capable GPU installed?";
			exit(1);
		}
	}

	cout << "starting on device #" <<  device_id << " at " << current << " with " << block_count << " blocks, each with " << THREADS_PER_BLOCK << " threads" << endl
	     << "if you ever want to pause the program then that's too bad, I can't be bothered to implement it" << endl << endl;

	size_t total_thread_count = block_count * THREADS_PER_BLOCK;
	size_t decode_base64_block_count = block_count * max_key_size/4;

	char* current_ptr_end = current_ptr + current_length;
	for (char* ptr = current_ptr; ptr != current_ptr_end; ++ptr) {
		*ptr = find(ALPHABET, ALPHABET + 65, *ptr) - ALPHABET;
	}

	/*/
	char* decrypted_base64;
	if (hipHostAlloc(&decrypted_base64, total_thread_count * CIPHERTEXT_LEN, hipHostMallocDefault) != hipSuccess) {
		cout << "hipHostAlloc failed";
		goto Err;
	}
	char* decrypted_base64_end = decrypted_base64 + total_thread_count * CIPHERTEXT_LEN;
	//*/
	/*/
	char* decoded_utf8;
	if (hipHostAlloc(&decoded_utf8, total_thread_count * UTF8_LEN, hipHostMallocDefault) != hipSuccess) {
		cout << "hipHostAlloc failed";
		goto Err;
	}
	char* decoded_utf8_end = decoded_utf8 + total_thread_count * UTF8_LEN;
	//*/
	char* keys;
	if (hipHostAlloc(&keys, total_thread_count * max_key_size, hipHostMallocWriteCombined) != hipSuccess) {
		cout << "hipHostAlloc failed";
		goto Err;
	}
	char* decode_success;
	if (hipHostAlloc(&decode_success, total_thread_count * max_key_size, hipHostMallocMapped) != hipSuccess) {
		cout << "hipHostAlloc failed";
		goto Err;
	}
	char* decode_success_end = decode_success + total_thread_count;
	char* decode_success_device = 0;
	if (hipHostGetDevicePointer((void**)&decode_success_device,(void*)decode_success,0) != hipSuccess) {
		cout << "hipHostGetDevicePointer failed";
		goto Err;
	}
	char* keys_device;
	if (hipMalloc(&keys_device, total_thread_count * max_key_size) != hipSuccess) {
		cout << "hipMalloc failed";
		goto Err;
	}
	char* decrypted_base64_device;
	if (hipMalloc(&decrypted_base64_device, total_thread_count * CIPHERTEXT_LEN) != hipSuccess) {
		cout << "hipMalloc failed";
		goto Err;
	}
	char* decoded_utf8_device;
	if (hipMalloc(&decoded_utf8_device, (total_thread_count * UTF8_LEN)) != hipSuccess) {
		cout << "hipMalloc failed";
		goto Err;
	}


	char* keys_buffer;
	if (hipHostAlloc(&keys_buffer, total_thread_count * max_key_size, hipHostMallocWriteCombined) != hipSuccess) {
		cout << "hipHostAlloc failed";
		goto Err;
	}
	char* decode_success_buffer;
	if (hipHostAlloc(&decode_success_buffer, total_thread_count * max_key_size, hipHostMallocMapped) != hipSuccess) {
		cout << "hipHostAlloc failed";
		goto Err;
	}
	char* decode_success_buffer_end = decode_success_buffer + total_thread_count;
	char* decode_success_buffer_device = 0;
	if (hipHostGetDevicePointer((void**)&decode_success_buffer_device,(void*)decode_success_buffer,0) != hipSuccess) {
		cout << "hipHostGetDevicePointer failed";
		goto Err;
	}
	char* keys_device_buffer;
	if (hipMalloc(&keys_device_buffer, total_thread_count * max_key_size) != hipSuccess) {
		cout << "hipMalloc failed";
		goto Err;
	}
	char* decrypted_base64_device_buffer;
	if (hipMalloc(&decrypted_base64_device_buffer, total_thread_count * CIPHERTEXT_LEN) != hipSuccess) {
		cout << "hipMalloc failed";
		goto Err;
	}
	char* decoded_utf8_device_buffer;
	if (hipMalloc(&decoded_utf8_device_buffer, (total_thread_count * UTF8_LEN)) != hipSuccess) {
		cout << "hipMalloc failed";
		goto Err;
	}
	/*bool* success_device_buffer;
	if (hipMalloc(&success_device_buffer, total_thread_count * sizeof(bool)) != hipSuccess) {
		cout << "hipMalloc failed";
		goto Err;
	}*/

	hipStream_t validate_stream, copy_stream;
	hipStreamCreate(&validate_stream);
	hipStreamCreate(&copy_stream);

	auto sync = [&] () {
		cuda_err = hipDeviceSynchronize();
		if (cuda_err != hipSuccess) {
			cout << endl << "hipDeviceSynchronize returned error after launching kernel: " << hipGetErrorString(cuda_err) << endl;
			hipDeviceReset();
			file.close();
			system("pause");
			exit(1);
		}
	};

	auto sync_stream = [&](hipStream_t stream) {
		cuda_err = hipStreamSynchronize(stream);
		if (cuda_err != hipSuccess) {
			cout << "hipDeviceSynchronize returned error after launching kernel: " << hipGetErrorString(cuda_err) << endl;
			hipDeviceReset();
			file.close();
			system("pause");
			exit(1);
		}
	};

	auto start = chrono::high_resolution_clock::now();
	auto decrypt_start = start;

	uint8_t i = 0;
	for (;;++i) {
		/* #region GENERATE */
		cout << "generating " << total_thread_count << " new keys, starting at ";
		
		char* current_ptr_end = current_ptr + current_length;
		for (char* ptr = current_ptr; ptr != current_ptr_end; ++ptr) {
			cout << ALPHABET[*ptr];
		}

		start = chrono::high_resolution_clock::now();
		for (size_t i = 0; i < total_thread_count * max_key_size; i += max_key_size) {
			char* key_ptr = keys + i;
			for (size_t j = 0; j < max_key_size; ++j) { key_ptr[j] = current_ptr[j]; }

			add_one(current_ptr, current_length);

			/*/
			char* key_ptr_end = key_ptr + max_key_size;
			for (char* ch = key_ptr; ch < key_ptr_end; ++ch) {
				if (*ch < 10) {cout << 0;}
				cout << (int)*ch << " ";
			}
			cout << " " << (int)current_length << endl;
			//*/
		}
		cout << endl << "generated new keys in " << (chrono::high_resolution_clock::now() - start) / chrono::milliseconds(1) << "ms, transferring keys to device" << endl;
		/* #endregion */

		/* #region TRANSFER */
		start = chrono::high_resolution_clock::now();
		if (hipMemcpyAsync(keys_device, keys, total_thread_count * max_key_size, hipMemcpyHostToDevice, copy_stream) != hipSuccess) {
			cout << "hipMalloc failed";
			goto Err;
		}
		sync_stream(copy_stream);

		cout << "transferred in " << (chrono::high_resolution_clock::now() - start) / chrono::milliseconds(1) << "ms" << endl;
		/* #endregion */

		sync();
		cout << "decrypted in " << (chrono::high_resolution_clock::now() - decrypt_start) / chrono::milliseconds(1) << "ms" << endl;

		/* #region DECODE */
		start = chrono::high_resolution_clock::now();
		decode_base64<<<decode_base64_block_count, THREADS_PER_BLOCK>>>(decrypted_base64_device, decoded_utf8_device);

		cuda_err = hipGetLastError();
		if (cuda_err != hipSuccess) {
			cout << "decode_base64 kernel launch failed: " << hipGetErrorString(cuda_err) << endl;
			exit(1);
		}

		cout << "decoded keys in " << (chrono::high_resolution_clock::now() - start) / chrono::milliseconds(1) << "ms, validating UTF-8 and decrypting buffered keys" << endl;
		/* #endregion */

		sync();

		/* #region VALIDATE START */
		start = chrono::high_resolution_clock::now();
		validate_utf8<<<block_count, THREADS_PER_BLOCK>>>(max_key_size, keys_device_buffer, decoded_utf8_device, decode_success_device);

		cuda_err = hipGetLastError();
		if (cuda_err != hipSuccess) {
			cout << "decode_base64 kernel launch failed: " << hipGetErrorString(cuda_err) << endl;
			exit(1);
		}
		/* #endregion */

		sync();

		/*if(hipMemcpyAsync(decode_success,success_device,total_thread_count,hipMemcpyDeviceToHost,copy_stream) != hipSuccess) {
			cout << "hipMemcpy failed" << endl;
			goto Err;
		}
		sync_stream(copy_stream);*/

		decrypt_start = chrono::high_resolution_clock::now();
		compute<<<block_count, THREADS_PER_BLOCK>>>(current_length, max_key_size, keys_device, decrypted_base64_device);

		cout << "validated keys in " << (chrono::high_resolution_clock::now() - start) / chrono::milliseconds(1) << "ms, processing on CPU" << endl;
		start = chrono::high_resolution_clock::now();

		/*/
		if (hipMemcpy(decoded_utf8, decoded_utf8_device, total_thread_count * UTF8_LEN, hipMemcpyDeviceToHost) != hipSuccess) {
			cout << "hipMemcpy failed";
			goto Err;
		}

		for (char* ptr = decoded_utf8; ptr < decoded_utf8_end; ptr += UTF8_LEN) {
			char* ptr_end = ptr + CIPHERTEXT_LEN;
			for (char* ch = ptr; ch < ptr_end; ++ch) {
				cout << *ch;
			}
			cout << endl;
		}
		//*/
		/*/
		if (hipMemcpy(decrypted_base64, decrypted_base64_device, total_thread_count * CIPHERTEXT_LEN, hipMemcpyDeviceToHost) != hipSuccess) {
			cout << "hipMemcpy failed";
			goto Err;
		}

		for (char* ptr = decrypted_base64; ptr < decrypted_base64_end; ptr += CIPHERTEXT_LEN) {
			char* ptr_end = ptr + CIPHERTEXT_LEN;
			for (char* ch = ptr; ch < ptr_end; ++ch) {
				cout << *ch;
			}
			cout << endl;
		}
		//*/

		cout << "copied in " << (chrono::high_resolution_clock::now() - start) / chrono::milliseconds(1) << "ms, processing results" << endl;
		start = chrono::high_resolution_clock::now();

		uint16_t found = 0;
		for (char* ptr = decode_success; ptr < decode_success_end; ptr+=max_key_size) {
			if (*ptr != 0) {
				found++;
				size_t i = ptr - decode_success;
				cout << endl << "DECODED SUCCESSFULLY WITH KEY ";
				for (char* ch = ptr; ch < ptr + current_length; ++ch) {
					cout << *ch;
					file << *ch;
				}
				cout << endl;
				file << ": ";

				char* decrypted_base64 = (char*)malloc(CIPHERTEXT_LEN);
				if (hipMemcpy(decrypted_base64, decrypted_base64_device + i * CIPHERTEXT_LEN, CIPHERTEXT_LEN, hipMemcpyDeviceToHost) != hipSuccess) {
					cout << "hipMemcpy failed";
					goto Err;
				}
				char* decoded_utf8 = (char*)malloc(UTF8_LEN);
				modp_b64_decode_h(decoded_utf8, decrypted_base64, CIPHERTEXT_LEN);

				for (char* ch = decoded_utf8; ch < decoded_utf8 + UTF8_LEN; ++ch) {
					file << *ch;
					cout << *ch;
				}
				file << endl;
				cout << endl << endl;

				free(decoded_utf8); free(decrypted_base64);
				*ptr = 0;
			}
		}

		cout << found << " matches found in " << (chrono::high_resolution_clock::now() - start) / chrono::milliseconds(1) << "ms" << endl;
		if (found > 0) {system("pause");}
		/* #endregion */

		swap((void**)&keys_device, (void**)&keys_device_buffer);
		swap((void**)&decrypted_base64_device, (void**)&decrypted_base64_device_buffer);
		swap((void**)&decoded_utf8_device, (void**)&decoded_utf8_device_buffer);
		
		swap((void**)&keys, (void**)&keys_buffer);
		swap((void**)&decode_success, (void**)&decode_success_buffer);
		swap((void**)&decode_success_device, (void**)&decode_success_buffer_device);

		//this_thread::sleep_for(chrono::milliseconds(5000));
	}

	exit(0);

Err:
	hipDeviceReset();
	file.close();
	system("pause");
	exit(1);
}